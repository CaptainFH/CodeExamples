#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<cstdlib>
#include<opencv4/opencv2/opencv.hpp>
#include<pcl/io/pcd_io.h>
#include<pcl/point_types.h>
#include<pcl/conversions.h>

__global__
void test(){
    printf("%d core is saying hi\n", threadIdx.x);
}

__global__ 
void pointCloudGenerator(float *disparity,float *resultX, float *resultY, float *resultZ, int length, float pixelAngleX, float pixelAngleY, int rowLength, float cameraRelativeDistance, float totalAngle, float angleDifPerPixel, float *leftCameraPosition){
    int dimension = blockIdx.x;
    int start = threadIdx.x;
    int currentPixelX;
    int currentPixelY;
    float angleX;
    float angleY;
    float angleXR;
    for(int i = start; i < length; i+=dimension){
        currentPixelY = i/rowLength;
        currentPixelX =  i%rowLength;
        angleX = totalAngle - (angleDifPerPixel*currentPixelX);
        angleY = totalAngle - (angleDifPerPixel*currentPixelY);
        angleXR = totalAngle - (angleDifPerPixel*(currentPixelX + disparity[i]));
        //Calculating angles is gonna be hard....
        if((angleX + angleXR) >= 180){
            break;
        }
        resultZ[i] = (cameraRelativeDistance*std::sin(angleX) * std::sin(angleXR)) / (std::sin(angleX+angleXR));
        resultX[i] = leftCameraPosition[0] + (resultZ[i] / std::tan(angleX));
        resultY[i] = leftCameraPosition[1] + (resultZ[i] / std::tan(angleY));
    }
}
__global__
void findDisparity(float *averagePixels1, float *averagePixels2, float *disparity, int length){
   int dimension = blockDim.x;
   int start = threadIdx.x;
   float similarity;
   float currentSimilarity;
   for(int i = start; i < length; i+=dimension){
        similarity = 100;
        disparity[i] = length -1;
        for(int j = start; j < length; j+=dimension){
             currentSimilarity = std::abs(averagePixels1[i] - averagePixels2[j]);
            if(std::abs(currentSimilarity) < std::abs(similarity)){
                similarity = currentSimilarity;
                disparity[i] = i - j;
            }
         }
   }
}
__global__
void blockMatching(float *pixels, float *averageMap, int length, int firstRow){
    int dimension = blockDim.x;
    int average;
    int start = threadIdx.x;
    for(int i = start; i < length; i+=dimension){
        average = 9;
        averageMap[i] = pixels[i];
        if (i > firstRow){//include the others
            averageMap[i] += pixels[i-dimension];
            if(i%dimension != 1){
                averageMap[i] += pixels[i-dimension+1];
            }else{
                average--;
            }
            if (i%dimension != 0){
                averageMap[i] += pixels[i-dimension-1];
            }else{
                average--;
            }
        }else{
            average-=3;
        }
        if(i%dimension != 1){
            averageMap[i] += pixels[i+1];
        }else{
            average--;
        }
        if (i%dimension != 0){
            averageMap[i] += pixels[i-1];
        }else{
             average--;
        }
        if(i+dimension > length){
            averageMap[i] += pixels[i+dimension];
            if (i%dimension != 0){
                averageMap[i] += pixels[i+dimension-1];
            }else{
                average--;
            }
            if(i%dimension != 1){
                averageMap[i] += pixels[i+dimension+1];
            }else{
                average--;
            }
        }else{
            average--;
        }
        averageMap[i] = averageMap[i] / average;
    }
}   

//Camera Calibration Values
float fov;
float cameraRelativeDistance;
float leftCameraPositionX;
float rightCameraPositionX;
float leftCameraPositionY;
float rightCameraPositionY;
///////////////////////////////////////////////////////////////



//separate the images into the values of the lines, based on the size of the blocks.
//Cuda malloc is used for global memory
//Local memory is used when using to many registers per thread - it's scope is per thread 
//L1 is the individual multiprocessor cache while L2 is the share memory - they are composed by the same bytes 
//- divided by us
//constant memory is good beacuse fast
//Texture memory is read only - but can be accessed by all threats 
//Shared memory is extremely fast used by a block of memory
//registers are fast- declared in a kernel.
//
//Matching is wack! just figure out how ya gonna be able to observe the second image's result in order to match it.
//Makit it one! To avoid memory leakage!!!!
//FIX TRIANGULATION BY MAKING SURE THE VALUES BETWEEN EACH PIXEL CORRESPOND TO THE PIXEL SIZE!!!!
int blockSize = 9;

int main (int argc, char** argv){
    //Extract the images from the input
    cv::Mat image = cv::imread(argv[1], cv::IMREAD_COLOR);
    cv::Mat image2 = cv::imread(argv[2], cv::IMREAD_COLOR);
    //Converts them to grayscale
    cv::cvtColor(image,image,cv::COLOR_RGB2GRAY);
    cv::cvtColor(image2,image2,cv::COLOR_RGB2GRAY);
    //Declaring neeeded memory
    float *imagePixels;
    float *image2Pixels;
    float *imagePixelsInGpu;
    float *image2PixelsInGpu;
    float *averageMap1;
    float *averageMap2;
    float *disparityMap;
    float *resultX;
    float *resultY;
    float *resultZ;
    //Allocating needed memory CPU
    int totalPixels = (image.cols * image.rows);
    imagePixels = (float*)malloc(sizeof(float) * totalPixels);
    image2Pixels = (float*)malloc(sizeof(float) * totalPixels);
    //Allocating needed memory GPU
    hipMalloc(&imagePixelsInGpu, sizeof(float) * totalPixels);
    hipMalloc(&image2PixelsInGpu, sizeof(float) * totalPixels);
    hipMalloc(&averageMap1, sizeof(float) * totalPixels);
    hipMalloc(&averageMap2, sizeof(float) * totalPixels);
    hipMalloc(&disparityMap, sizeof(float)* totalPixels);
    hipMalloc(&resultX, sizeof(float)* totalPixels);
    hipMalloc(&resultY, sizeof(float)* totalPixels);
    hipMalloc(&resultZ, sizeof(float)* totalPixels);
    //separating the image into a float array
    int blocksDef = image.cols/blockSize;
    for (int i = 0; i < image.cols; i++){
        for (int j = 0; j  < image.rows; j++){
            imagePixels[i] = (float)image.at<short>(i,j);
            image2Pixels[i] = (float)image2.at<short>(i,j);
        }
    }
    std::cout << 3/5 << std:: endl;
    blockMatching<<<1,image.cols>>>(imagePixelsInGpu, averageMap1, totalPixels, image.rows);
    hipDeviceSynchronize();
    blockMatching<<<1,image2.cols>>>(image2PixelsInGpu,averageMap2,totalPixels, image2.rows);
    hipDeviceSynchronize();
    findDisparity(averageMap1,averageMap2,disparityMap,totalPixels);
    hipDeviceSynchronize();
    //Divde height of the image by numblocks for thread blocks 
    //Divide widgth by blocks for the number of threads.
    //hipMalloc(*image1AsVector, sizeof(float) *(image.rows * image.cols),);

    test<<<1,1>>>();
    pcl::PointCloud<pcl::_PointXYZ> result;
    pcl::PointXYZ tempPoint;
    for(int i = 0; i < totalPixels; i++){
        tempPoint.x = resultX[i];
        tempPoint.y = resultY[i];
        tempPoint.z = resultZ[i];
        result.points.push_back(tempPoint);
    }

    pcl::PCLPointCloud2 msg;
    pcl::toPCLPointCloud2(result,msg);

    std::cout << result << std::endl;

    cv::imshow(argv[1], image);  
    cv::waitKey();
    free(imagePixels);
    free(image2Pixels);
    hipFree(imagePixelsInGpu);
    hipFree(image2PixelsInGpu);
    hipFree(averageMap1);
    hipFree(averageMap2);
    hipFree(disparityMap);
    hipFree(resultX);
    hipFree(resultY);
    hipFree(resultZ);
    return 0;
}

cv::Mat parallelMatcher(cv::Mat imageOne, cv::Mat imageTwo, int blockSize){
    cv::Mat result; 
    cv::Mat imageOneGPU, imageTwoGPU;
    //hipMalloc(&imageOneGPU,sizeof(imageOne));
    //hipMalloc(&imageTwoGPU,sizeof(imageTwo));

    return result;
}