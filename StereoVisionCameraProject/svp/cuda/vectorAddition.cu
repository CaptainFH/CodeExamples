
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<iostream>

#define N 10000000
__global__
void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    //Allocater GPU memory
    hipMalloc((void**)&d_a, sizeof(float)*N);

    //Copy data from cpu to GPU
    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) 
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    //cudaProfilerStop();
    // Main function
    vector_add<<<1,1>>>(out, d_a, b, N);
    free(a);
    free(b);
    free(out);
    hipFree(d_a);
    printf("worked");
    return 0;
}